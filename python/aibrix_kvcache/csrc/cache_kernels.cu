#include "hip/hip_runtime.h"
// Adapted from vLLM
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#ifdef USE_ROCM
#include "quantization/fp8/amd/quant_utils.cuh"
#else
#include "quantization/fp8/nvidia/quant_utils.cuh"
#endif

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

namespace aibrix {

template <typename TPtr, typename TTensor>
TPtr *get_device_ptr(TTensor &tensor) {
  torch::Device device = tensor.device();
  bool is_pinned = tensor.is_pinned();

  if (device.is_cuda()) {
    return reinterpret_cast<TPtr *>(tensor.data_ptr());
  } else if (is_pinned) {
    void *ptr;
    hipHostGetDevicePointer(&ptr, static_cast<void *>(tensor.data_ptr()), 0);
    return static_cast<TPtr *>(ptr);
  }

  TORCH_CHECK(false, "Tensor must be on GPU or be pinned");
  return nullptr;
}

template <typename TTensor>
torch::Tensor get_device_ptrs(const std::vector<TTensor> &tensors) {
  // Create a vector to store the GPU memory pointers
  std::vector<void *> data_ptrs;
  data_ptrs.reserve(tensors.size());

  // Extract data pointers
  for (const auto &tensor : tensors) {
    data_ptrs.push_back(get_device_ptr<void *>(tensor));
  }

  torch::Tensor gpu_data_ptrs =
      torch::from_blob(data_ptrs.data(),
                       {static_cast<int64_t>(data_ptrs.size())}, torch::kInt64)
          .to(torch::kCUDA);

  return gpu_data_ptrs;
}

__device__ __forceinline__ int64_t
get_kv_cache_offset(const int64_t kv_type, const int64_t num_blocks,
                    const int64_t block_size, const int64_t embed_dim,
                    const int64_t slot_idx, const int64_t scalar_offset) {
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;
  return kv_type * num_blocks * block_size * embed_dim +
         block_idx * block_size * embed_dim + block_offset * embed_dim +
         scalar_offset;
}

__device__ __forceinline__ int64_t get_offload_kv_cache_offset_lcnd(
    const int64_t kv_type, const int64_t layer_idx, const int64_t block_size,
    const int64_t num_layers, const int64_t embed_dim, const int64_t token_idx,
    const int64_t scalar_offset) {
  const int64_t block_offset = token_idx % block_size;
  return layer_idx * 2 * block_size * embed_dim +
         kv_type * block_size * embed_dim + block_offset * embed_dim +
         scalar_offset;
}

__device__ __forceinline__ int64_t get_offload_kv_cache_offset_ncld(
    const int64_t kv_type, const int64_t layer_idx, const int64_t block_size,
    const int64_t num_layers, const int64_t embed_dim, const int64_t token_idx,
    const int64_t scalar_offset) {
  const int64_t block_offset = token_idx % block_size;
  return block_offset * 2 * num_layers * embed_dim +
         kv_type * num_layers * embed_dim + layer_idx * embed_dim +
         scalar_offset;
}

enum class KVCacheOffloadLayout {
  kLCND = 1,
  kNCLD,
};

/*
 * Template arguments:
 * - TOnload: True if offload_kv_cache to kv_cache.
 * - TLayout: The layout of offload_kv_cache.
 * Args:
 * - offload_kv_cache: Supports LCND and NCLD layouts.
 *                     LCND: [num_blocks, num_layers, 2, block_size, dim]
 *                     NCLD: [num_blocks, block_size, 2, num_layers, dim]
 * - kv_cache: Supports both [num_layers, 2, num_blocks, block_size, num_heads,
 * head_size] and [num_layers, 2, num_blocks, block_size * num_heads *
 * head_size]
 * - slot_mapping: [num_tokens]
 */
template <typename scalar_t, typename cache_t, vllm::Fp8KVCacheDataType kv_dt,
          bool TOnload, KVCacheOffloadLayout TLayout>
__global__ void reshape_and_cache_multi_layer_kernel(
    scalar_t **__restrict__ offload_kv_cache,
    const int64_t offload_kv_cache_block_size, cache_t **__restrict__ kv_cache,
    const int64_t kv_cache_block_size, const int64_t kv_cache_num_blocks,
    const int64_t *__restrict__ slot_mapping, const int64_t num_layers,
    const int64_t embed_dim,
    const float **k_scales, // Scaling factor for keys
    const float **v_scales  // Scaling factor for values
) {
  const int64_t token_idx = blockIdx.x;
  const int64_t layer_idx = blockIdx.y;
  const int64_t kv_type = blockIdx.z;
  const int64_t tid = threadIdx.x;
  const int64_t num_threads = blockDim.x;

  const int64_t slot_idx = slot_mapping[token_idx];
  if (slot_idx < 0)
    return;

  const int64_t offload_kv_cache_block_idx =
      token_idx / offload_kv_cache_block_size;

  scalar_t *offload_kv_cache_block =
      offload_kv_cache[offload_kv_cache_block_idx];
  cache_t *kv_cache_layer = kv_cache[layer_idx];
  const float *k_scale = k_scales[layer_idx];
  const float *v_scale = v_scales[layer_idx];

  // Copy data between kv_cache and offload_kv_cache
  for (int i = tid; i < embed_dim; i += num_threads) {
    int64_t offload_kv_cache_offset = 0;
    if constexpr (TLayout == KVCacheOffloadLayout::kLCND) {
      offload_kv_cache_offset = get_offload_kv_cache_offset_lcnd(
          kv_type, layer_idx, offload_kv_cache_block_size, num_layers,
          embed_dim, token_idx, i);
    } else {
      offload_kv_cache_offset = get_offload_kv_cache_offset_ncld(
          kv_type, layer_idx, offload_kv_cache_block_size, num_layers,
          embed_dim, token_idx, i);
    }

    int64_t kv_cache_offset =
        get_kv_cache_offset(kv_type, kv_cache_num_blocks, kv_cache_block_size,
                            embed_dim, slot_idx, i);

    if (TOnload) { // true: offload_kv_cache to kv_cache
      kv_cache_layer[kv_cache_offset] =
          vllm::fp8::scaled_convert<cache_t, scalar_t, kv_dt>(
              offload_kv_cache_block[offload_kv_cache_offset],
              (kv_type == 0) ? *k_scale : *v_scale);
    } else { // false: kv_cache to offload_kv_cache
      offload_kv_cache_block[offload_kv_cache_offset] =
          vllm::fp8::scaled_convert<scalar_t, cache_t, kv_dt>(
              kv_cache_layer[kv_cache_offset],
              (kv_type == 0) ? *k_scale : *v_scale);
    }
  }
}

/*
 * Template arguments:
 * - TOnload: True if offload_kv_cache to kv_cache.
 * - TLayout: The layout of offload_kv_cache.
 * Args:
 * - offload_kv_cache: Supports LCND and NCLD layouts.
 *                     LCND: [num_blocks, num_layers, 2, block_size, dim]
 *                     NCLD: [num_blocks, block_size, 2, num_layers, dim]
 * - kv_cache: Supports both [num_layers, 2, num_blocks, block_size, num_heads,
 * head_size] and [num_layers, 2, num_blocks, block_size * num_heads *
 * head_size]
 * - slot_mapping: [num_tokens]
 */
template <typename vec_t, bool TOnload, KVCacheOffloadLayout TLayout>
__global__ void reshape_and_cache_multi_layer_vec_kernel(
    vec_t **__restrict__ offload_kv_cache,
    const int64_t offload_kv_cache_block_size, vec_t **__restrict__ kv_cache,
    const int64_t kv_cache_block_size, const int64_t kv_cache_num_blocks,
    const int64_t *__restrict__ slot_mapping, const int64_t num_layers,
    const int64_t num_vecs) {
  const int64_t token_idx = blockIdx.x;
  const int64_t layer_idx = blockIdx.y;
  const int64_t kv_type = blockIdx.z;
  const int64_t tid = threadIdx.x;
  const int64_t num_threads = blockDim.x;

  const int64_t slot_idx = slot_mapping[token_idx];
  if (slot_idx < 0)
    return;

  const int64_t offload_kv_cache_block_idx =
      token_idx / offload_kv_cache_block_size;

  vec_t *offload_kv_cache_block = offload_kv_cache[offload_kv_cache_block_idx];
  vec_t *kv_cache_layer = kv_cache[layer_idx];

  // Copy data between kv_cache and offload_kv_cache
  for (int i = tid; i < num_vecs; i += num_threads) {
    int64_t offload_kv_cache_offset = 0;
    if constexpr (TLayout == KVCacheOffloadLayout::kLCND) {
      offload_kv_cache_offset = get_offload_kv_cache_offset_lcnd(
          kv_type, layer_idx, offload_kv_cache_block_size, num_layers, num_vecs,
          token_idx, i);
    } else {
      offload_kv_cache_offset = get_offload_kv_cache_offset_ncld(
          kv_type, layer_idx, offload_kv_cache_block_size, num_layers, num_vecs,
          token_idx, i);
    }

    int64_t kv_cache_offset =
        get_kv_cache_offset(kv_type, kv_cache_num_blocks, kv_cache_block_size,
                            num_vecs, slot_idx, i);

    if (TOnload) { // true: offload_kv_cache to kv_cache
      kv_cache_layer[kv_cache_offset] =
          offload_kv_cache_block[offload_kv_cache_offset];
    } else { // false: kv_cache to offload_kv_cache
      offload_kv_cache_block[offload_kv_cache_offset] =
          kv_cache_layer[kv_cache_offset];
    }
  }
}

// KV_T is the data type of offload kv cache.
// CACHE_T is the stored data type of kv cache.
// KV_DTYPE is the real data type of kv cache.
// onload_const: true if offload_kv_cache to kv_cache
// layout_const: LCND, NCLD
#define CALL_RESHAPE_AND_CACHE_MULTI_LAYER(KV_T, CACHE_T, KV_DTYPE)            \
  reshape_and_cache_multi_layer_kernel<KV_T, CACHE_T, KV_DTYPE, onload_const,  \
                                       layout_const>                           \
      <<<grid, block, 0, stream>>>(                                            \
          reinterpret_cast<KV_T **>(offload_kv_cache_ptrs.data_ptr()),         \
          offload_kv_cache_block_size,                                         \
          reinterpret_cast<CACHE_T **>(kv_cache_ptrs.data_ptr()), block_size,  \
          kv_cache_num_blocks, slot_mapping.data_ptr<int64_t>(), num_layers,   \
          embed_dim,                                                           \
          reinterpret_cast<const float **>(k_scale_ptrs.data_ptr()),           \
          reinterpret_cast<const float **>(v_scale_ptrs.data_ptr()));

#define CALL_RESHAPE_AND_CACHE_MULTI_LAYER_VEC                                 \
  reshape_and_cache_multi_layer_vec_kernel<vec_t, onload_const, layout_const>  \
      <<<grid, block, 0, stream>>>(                                            \
          reinterpret_cast<vec_t **>(offload_kv_cache_ptrs.data_ptr()),        \
          offload_kv_cache_block_size,                                         \
          reinterpret_cast<vec_t **>(kv_cache_ptrs.data_ptr()), block_size,    \
          kv_cache_num_blocks, slot_mapping.data_ptr<int64_t>(), num_layers,   \
          num_vecs);

#define DISPATCH_RESHAPE_AND_CACHE_MULTI_LAYER_BY_KV_CACHE_DTYPE               \
  DISPATCH_BY_KV_CACHE_DTYPE(kv_caches[0].dtype(), kv_cache_dtype,             \
                             CALL_RESHAPE_AND_CACHE_MULTI_LAYER);

#define DISPATCH_RESHAPE_AND_CACHE_MULTI_LAYER_BY_ONLOAD_AND_LAYOUT(           \
    ONLOAD_T, LAYOUT_T, FN)                                                    \
  if (LAYOUT_T == aibrix::KVCacheOffloadLayout::kLCND) {                       \
    const auto layout_const = aibrix::KVCacheOffloadLayout::kLCND;             \
    if (ONLOAD_T) {                                                            \
      const auto onload_const = true;                                          \
      FN;                                                                      \
    } else {                                                                   \
      const auto onload_const = false;                                         \
      FN;                                                                      \
    }                                                                          \
  } else {                                                                     \
    const auto layout_const = aibrix::KVCacheOffloadLayout::kNCLD;             \
    if (ONLOAD_T) {                                                            \
      const auto onload_const = true;                                          \
      FN;                                                                      \
    } else {                                                                   \
      const auto onload_const = false;                                         \
      FN;                                                                      \
    }                                                                          \
  }

void reshape_and_cache_multi_layer_impl(
    const std::vector<torch::Tensor> &offload_kv_cache_blocks, // [num_blocks]
    const std::vector<torch::Tensor> &kv_caches,               // [num_layers]
    torch::Tensor &slot_mapping,                               // [num_tokens]
    const int64_t block_size, const std::string &kv_cache_dtype,
    const std::vector<torch::Tensor> &k_scales,
    const std::vector<torch::Tensor> &v_scales, bool onload,
    const std::string &layout_str) {
  const auto layout = (layout_str == "LCND")
                          ? aibrix::KVCacheOffloadLayout::kLCND
                          : aibrix::KVCacheOffloadLayout::kNCLD;
  const int64_t num_tokens = slot_mapping.size(0);
  torch::IntArrayRef kv_cache_shape = kv_caches[0].sizes();
  int64_t embed_dim;
  if (kv_cache_shape.size() == 3) {
    // [2, num_blocks, block_size * num_heads * head_size]
    const int64_t block_dim = kv_caches[0].stride(1);
    embed_dim = block_dim / block_size;
  } else {
    // [2, num_blocks, block_size, num_heads, head_size]
    embed_dim = kv_caches[0].stride(2);
  }

  torch::IntArrayRef offload_kv_cache_block_shape =
      offload_kv_cache_blocks[0].sizes();
  const int64_t offload_kv_cache_block_size =
      (layout == aibrix::KVCacheOffloadLayout::kLCND)
          ? offload_kv_cache_block_shape[2]
          : offload_kv_cache_block_shape[0];
  const int64_t offload_kv_cache_num_layers =
      (layout == aibrix::KVCacheOffloadLayout::kLCND)
          ? offload_kv_cache_block_shape[0]
          : offload_kv_cache_block_shape[2];

  TORCH_CHECK(num_tokens ==
              offload_kv_cache_blocks.size() * offload_kv_cache_block_size);

  const int64_t num_layers = kv_caches.size();
  TORCH_CHECK(num_layers == offload_kv_cache_num_layers);

  // Assume all layers have the same shape
  for (int64_t i = 0; i < num_layers; i++) {
    TORCH_CHECK(kv_cache_shape == kv_caches[i].sizes());
  }

  const int64_t kv_cache_num_blocks = kv_cache_shape[1];

  torch::Tensor offload_kv_cache_ptrs =
      aibrix::get_device_ptrs(offload_kv_cache_blocks);
  torch::Tensor kv_cache_ptrs = aibrix::get_device_ptrs(kv_caches);
  torch::Tensor k_scale_ptrs = aibrix::get_device_ptrs(k_scales);
  torch::Tensor v_scale_ptrs = aibrix::get_device_ptrs(v_scales);

  const at::cuda::OptionalCUDAGuard device_guard(device_of(kv_caches[0]));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(num_tokens, num_layers, 2);
  if (kv_cache_dtype == "auto") {
    auto element_size = kv_caches[0].element_size();
    using vec_t = __int128_t;
    const int64_t num_vecs = embed_dim / sizeof(vec_t) * element_size;
    dim3 block(std::min(num_vecs, static_cast<int64_t>(128)));
    DISPATCH_RESHAPE_AND_CACHE_MULTI_LAYER_BY_ONLOAD_AND_LAYOUT(
        onload, layout, CALL_RESHAPE_AND_CACHE_MULTI_LAYER_VEC);
  } else {
    dim3 block(std::min(embed_dim, static_cast<int64_t>(512)));
    DISPATCH_RESHAPE_AND_CACHE_MULTI_LAYER_BY_ONLOAD_AND_LAYOUT(
        onload, layout,
        DISPATCH_RESHAPE_AND_CACHE_MULTI_LAYER_BY_KV_CACHE_DTYPE);
  }
}
} // namespace aibrix

void reshape_and_cache_multi_layer(
    const std::vector<torch::Tensor> &offload_kv_cache_blocks, // [num_blocks]
    const std::vector<torch::Tensor> &kv_caches,               // [num_layers]
    torch::Tensor &slot_mapping,                               // [num_tokens]
    const int64_t block_size, const std::string &kv_cache_dtype,
    const std::vector<torch::Tensor> &k_scales,
    const std::vector<torch::Tensor> &v_scales, const std::string &layout_str) {
  aibrix::reshape_and_cache_multi_layer_impl(
      offload_kv_cache_blocks, kv_caches, slot_mapping, block_size,
      kv_cache_dtype, k_scales, v_scales, true, layout_str);
}

void reshape_and_offload_multi_layer(
    const std::vector<torch::Tensor> &offload_kv_cache_blocks, // [num_blocks]
    const std::vector<torch::Tensor> &kv_caches,               // [num_layers]
    torch::Tensor &slot_mapping,                               // [num_tokens]
    const int64_t block_size, const std::string &kv_cache_dtype,
    const std::vector<torch::Tensor> &k_scales,
    const std::vector<torch::Tensor> &v_scales, const std::string &layout_str) {
  aibrix::reshape_and_cache_multi_layer_impl(
      offload_kv_cache_blocks, kv_caches, slot_mapping, block_size,
      kv_cache_dtype, k_scales, v_scales, false, layout_str);
}
